#include "hip/hip_runtime.h"
#include <cstdint>
#include <hipcub/hipcub.hpp>
#include <vector>
struct BitMask128 {
  uint64_t x, y;
};
typedef uint32_t ShapeMask;

struct GpuStep {
  BitMask128 *space;
  ShapeMask *unused;
  uint64_t *outpos;
  int *parent;
  int *order;
  int bufsize;
};

struct GpuTempSpace {
  int n_shapes;
  BitMask128 *imagelist;
  int *imgpos;
  int *counts;
  int *tmp_sum;
  size_t tmp_sum_bytes;
  uint64_t *out_pos_size;
};

static_assert(sizeof(ulonglong2) == sizeof(BitMask128), "??");

__device__ int ffs128_gpu(ulonglong2 a) {
  uint64_t nz = a.x;
  int off = 0;
  if (a.x == 0) {
    nz = a.y;
    off = 64;
  }
  return __ffsll(nz) + off - 1;
}

__device__ bool can_fit_gpu(ulonglong2 space, ulonglong2 image) {
  return (space.x & image.x) == image.x && (space.y & image.y) == image.y;
}

__global__ void compute_output_size_gpu(
  ulonglong2 *imagelist, int *imgpos, int n_shapes,
  ulonglong2 *cur_space, ShapeMask *cur_unused,
  int n_cur, int *out_count
) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= n_cur) {
    return;
  }
  ulonglong2 space = cur_space[idx];
  ShapeMask unused = cur_unused[idx];
  int choose = ffs128_gpu(space);
  int cntnext = 0;
  for (int i = 0; i < n_shapes; i++) {
    if (unused >> i & 1) {
      int end = imgpos[choose * n_shapes + i + 1];
      for (int j = imgpos[choose * n_shapes + i]; j < end; j++) {
        ulonglong2 m = imagelist[j];
        if (can_fit_gpu(space, m)) {
          cntnext++;
        }
      }
    }
  }
  out_count[idx] = cntnext;
  printf("out_count[%d]=%d\n", idx, cntnext);
}

__global__ void output_next_gpu(
  ulonglong2 *imagelist, int *imgpos, int n_shapes,
  ulonglong2 *cur_space, ShapeMask *cur_unused, uint64_t *cur_outpos,
  ulonglong2 *next_space, ShapeMask *next_unused, int *next_parent,
  int from, int to
) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= to - from) {
    return;
  }
  idx += from;
  ulonglong2 space = cur_space[idx];
  ShapeMask unused = cur_unused[idx];
  int choose = ffs128_gpu(space);
  int to_insert = idx == 0 ? 0 : cur_outpos[idx-1];
  for (int i = 0; i < n_shapes; i++) {
    if (unused >> i & 1) {
      int end = imgpos[choose * n_shapes + i + 1];
      for (int j = imgpos[choose * n_shapes + i]; j < end; j++) {
        ulonglong2 m = imagelist[j];
        if (can_fit_gpu(space, m)) {
          next_space[to_insert] = make_ulonglong2(space.x ^ m.x, space.y ^ m.y);
          next_unused[to_insert] = unused - (1U<<i);
          next_parent[to_insert] = idx;
          to_insert++;
        }
      }
    }
  }
}

__global__ void search_range_gpu(
  uint64_t *cur_outpos, int from, int to, int buf_size,
  uint64_t out_pos_size[2]
) {
  uint64_t base = from == 0 ? 0 : cur_outpos[from-1];
  int lo = from, hi = to;
  while (lo < hi) {
    int mid = (lo + hi + 1) / 2;
    if (cur_outpos[mid-1] - base > buf_size) {
      hi = mid - 1;
    } else {
      lo = mid;
    }
  }
  out_pos_size[0] = lo;
  if (lo == from) {
    out_pos_size[1] = 0;
  } else {
    out_pos_size[1] = cur_outpos[lo-1] - base;
  }
}

void runGpu(std::vector<GpuStep> g, GpuTempSpace gs) {
  int lv = 0;
  int end = 1;
  int depth = g.size();
  std::vector<int> segStart(depth);
  std::vector<int> segAll(depth);
  std::vector<uint64_t> outposPrev(depth);
  compute_output_size_gpu<<<4000,256>>>((ulonglong2 *) gs.imagelist, gs.imgpos, gs.n_shapes,
    (ulonglong2 *) g[lv].space, g[lv].unused, end, gs.counts);
  hipcub::DeviceScan::InclusiveSum(gs.tmp_sum, gs.tmp_sum_bytes, gs.counts, g[lv].outpos, end);
  segStart[lv] = 0;
  segAll[lv] = end;
  outposPrev[lv] = 0;
  while (lv < depth) {
    int start = segStart[lv];
    end = segAll[lv];
    uint64_t out_pos_size[2] = {0};
    search_range_gpu<<<1,1>>>(g[lv].outpos, start, end, g[lv].bufsize, gs.out_pos_size);
    hipMemcpy(out_pos_size, gs.out_pos_size, sizeof(uint64_t[2]), hipMemcpyDeviceToHost);
    end = out_pos_size[0];
    uint64_t outpos = out_pos_size[1];
    printf("lv.%d start=%d end=%d outpos=%lld\n", lv, start, end, outpos);
    if (end <= start) {
      printf("not enough output space\n");
      return;
    }
    segStart[lv] = end;
    uint64_t outsize = outpos - outposPrev[lv];
    outposPrev[lv] = outpos;
    if (outsize > 0 && lv < depth-1) {
      output_next_gpu<<<4000,256>>>((ulonglong2 *)gs.imagelist, gs.imgpos, gs.n_shapes,
        (ulonglong2 *)g[lv].space, g[lv].unused, g[lv].outpos,
        (ulonglong2 *)g[lv+1].space, g[lv+1].unused, g[lv+1].parent,
        start, end);
      lv++;
      segStart[lv] = 0;
      end = segAll[lv] = outsize;
      compute_output_size_gpu<<<4000,256>>>((ulonglong2 *) gs.imagelist, gs.imgpos, gs.n_shapes,
        (ulonglong2 *) g[lv].space, g[lv].unused, end, gs.counts);
      hipcub::DeviceScan::InclusiveSum(gs.tmp_sum, gs.tmp_sum_bytes, gs.counts, g[lv].outpos, end);
    } else {
      if (lv == depth-1) {
        printf("ans=%lld\n", outsize);
      }
      while (lv >= 0 && segStart[lv] == segAll[lv]) {
        outposPrev[lv] = 0;
        lv--;
      }
      if (lv == -1) return;
    }
  }
}

int main() {
  FILE *fin = fopen("e.txt", "r");
  if (!fin) {
    printf("no file\n");
    return 1;
  }
  int n_shapes = 0;
  int n_holes = 0;
  char buf[300];
  fscanf(fin, "%d %d", &n_shapes, &n_holes);
  if (n_shapes <= 0 || n_holes <= 0) {
    printf("number of shapes and holes must be > 0\n");
    return 1;
  }
  std::vector<std::vector<BitMask128>> shapelist(128 * n_shapes);
  for (int i = 0; i < n_shapes; i++) {
    int listlen = 0;
    fscanf(fin, "%d", &listlen);
    fgets(buf, 100, fin);
    for (int j = 0; j < listlen; j++) {
      fgets(buf, 200, fin);
      BitMask128 bm = {0, 0};
      int least = -1;
      for (int k = 0; k < n_holes; k++) {
        if (buf[k] == '1') {
          if (least == -1) least = k;
          if (k < 64) bm.x += 1ULL<<k;
          else bm.y += 1ULL<<(k-64);
        }
      }
      if (least != -1) {
        shapelist[least * n_shapes + i].push_back(bm);
      }
    }
  }
  std::vector<BitMask128> flatShapelist;
  std::vector<int> imgpos(shapelist.size() + 1);
  for (int i = 0; i < shapelist.size(); i++) {
    imgpos[i+1] = imgpos[i] + shapelist[i].size();
    for (BitMask128 bm : shapelist[i]) {
      flatShapelist.push_back(bm);
    }
  }

  int bufsize = 5;
  GpuTempSpace gs;
  gs.n_shapes = n_shapes;
  hipMalloc(&gs.imagelist, sizeof(BitMask128) * flatShapelist.size());
  hipMalloc(&gs.imgpos, sizeof(int) * imgpos.size());
  hipMalloc(&gs.counts, sizeof(int) * bufsize);
  uint64_t *dummy = nullptr;
  hipcub::DeviceScan::InclusiveSum(NULL, gs.tmp_sum_bytes, gs.counts, dummy, bufsize);
  hipMalloc(&gs.tmp_sum, gs.tmp_sum_bytes);
  hipMalloc(&gs.out_pos_size, sizeof(uint64_t) * 2);
  printf("prefix sum tmp %lld bytes\n", gs.tmp_sum_bytes);
  hipMemcpy(gs.imagelist, flatShapelist.data(), flatShapelist.size() * sizeof(BitMask128), hipMemcpyHostToDevice);
  hipMemcpy(gs.imgpos, imgpos.data(), imgpos.size() * sizeof(int), hipMemcpyHostToDevice);

  std::vector<GpuStep> g(n_shapes);
  for (int i = 0; i < n_shapes; i++) {
    hipMalloc(&g[i].space, sizeof(BitMask128) * bufsize);
    hipMalloc(&g[i].unused, sizeof(ShapeMask) * bufsize);
    hipMalloc(&g[i].outpos, sizeof(uint64_t) * bufsize);
    hipMalloc(&g[i].parent, sizeof(int) * bufsize);
    hipMalloc(&g[i].order, sizeof(int) * bufsize);
    g[i].bufsize = bufsize;
  }

  BitMask128 space = {0, 0};
  for (int k = 0; k < n_holes; k++) {
    if (k < 64) space.x += 1ULL<<k;
    else space.y += 1ULL<<(k-64);
  }
  ShapeMask unused = (1U<<(n_shapes-1)) * 2 - 1;
  hipMemcpy(g[0].space, &space, sizeof(BitMask128), hipMemcpyHostToDevice);
  hipMemcpy(g[0].unused, &unused, sizeof(ShapeMask), hipMemcpyHostToDevice);
  runGpu(g, gs);
  printf("%s\n", hipGetErrorString(hipDeviceSynchronize()));
}
