#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
__global__ void somecuda(int n, float *gg) {
  int tid=blockDim.x*blockIdx.x+threadIdx.x;
  for(int i=tid;i<n;i+=blockDim.x*gridDim.x) {
    gg[i] = gg[i]*gg[i];
  }
}

struct preCudaInit {
  int numDevices;
  std::vector<hipCtx_t> ctx;
  preCudaInit() {
    numDevices = 0;
    if (hipInit(0) != hipSuccess)
      std::cerr << "Error: CUDA initialization failed." << std::endl;
    else if (hipGetDeviceCount(&numDevices) != hipSuccess) 
      std::cerr << "Error: Cannot get device count." << '\n';
    else {
      ctx.resize(numDevices);
      for (int i = 0; i < numDevices; i++) {
        hipDevice_t dev;
        hipDeviceptr_t ptr;
        if (hipDeviceGet(&dev, i) != hipSuccess)
          std::cerr << "Error: Cannot get device " << i << "." << std::endl;
        else if (hipCtxCreate(&ctx[i], 0, dev) != hipSuccess)
          std::cerr << "Error: CUDA context creation failed." << std::endl;
        else if (hipMalloc(&ptr, 100) != hipSuccess)
          std::cerr << "Error: Cannot allocate memory with CUDA Driver API." << std::endl;
        else if (hipFree(ptr) != hipSuccess)
          std::cerr << "Error: Cannot deallocate memory with CUDA Driver API." << std::endl;
      }
    }
  }
  ~preCudaInit() {
    for (int i = 0; i < numDevices; i++) {
      hipDevice_t dev;
      if (hipDeviceGet(&dev, 0) != hipSuccess)
        std::cerr << "Error: Cannot get device " << i << "." << std::endl;
      else if (hipCtxDestroy(ctx[i]) != hipSuccess)
        std::cerr << "Error: CUDA finalization failed." << std::endl;
    }
  }
};
static preCudaInit cucu;

void ss() {
  int *leak;
  hipMalloc(&leak, 128);
}
