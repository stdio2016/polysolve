
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
__global__ void somecuda(int n, float *gg) {
  int tid=blockDim.x*blockIdx.x+threadIdx.x;
  for(int i=tid;i<n;i+=blockDim.x*gridDim.x) {
    gg[i] = gg[i]*gg[i];
  }
}

struct preCudaInit {
  int numDevices;
  std::vector<int> devices;
  preCudaInit() {
    numDevices = 0;
    if (hipGetDeviceCount(&numDevices) != hipSuccess) 
      std::cerr << "Error: Cannot get device count." << '\n';
    else {
      for (int i = 0; i < numDevices; i++) {
        if (hipSetDevice(i) != hipSuccess)
          std::cerr << "Error: Cannot switch to device " << i << ".\n";
        else if (hipFreeArray(nullptr) != hipSuccess)
          std::cerr << "Error: CUDA context initialization on device " << i << " failed.\n";
        else
          devices.push_back(i);
      }
    }
  }
  ~preCudaInit() {
    ;
  }
};
preCudaInit cucu;

void ss() {
  int *leak;
  hipMalloc(&leak, 128);
}
