
#include <hip/hip_runtime.h>
__global__ void somecuda(int n, float *gg) {
  int tid=blockDim.x*blockIdx.x+threadIdx.x;
  for(int i=tid;i<n;i+=blockDim.x*gridDim.x) {
    gg[i] = gg[i]*gg[i];
  }
}

__shared__ int a;
__device__ int r;
